
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void sort(int *key,int *bucket, int n, int range) {
 int i = blockIdx.x * blockDim.x + threadIdx.x;
 for(int j=0,k=0; k<=i; j++) {
   key[i]=j;
   __syncthreads();
   k+=bucket[j];
   __syncthreads();
  }
}

__global__ void bucket1(int *bucket){
 int i = blockIdx.x * blockDim.x + threadIdx.x;
 bucket[i] = 0;
}

__global__ void bucket2(int *key, int *bucket){
 int i = blockIdx.x * blockDim.x + threadIdx.x;
 atomicAdd(&bucket[key[i]],1);
}

int main() {
 int n = 50;
 int range = 5;
 int *key, *bucket;
 hipMallocManaged(&key,n*sizeof(int));
 hipMallocManaged(&bucket,range*sizeof(int));
 for (int i=0; i<n; i++) {
   key[i] = rand() % range;
   printf("%d ",key[i]);
  }
 printf("\n");

bucket1<<<1,range>>>(bucket);
bucket2<<<1,n>>>(key,bucket);
sort<<<1,n>>>(key, bucket, n, range);
hipDeviceSynchronize();

 for (int i=0; i<n; i++) {
   printf("%d ",key[i]);
  }
 printf("\n");
 hipFree(key);
 hipFree(bucket);
}
